#include "hip/hip_runtime.h"
/*
 * BoostintSolver.cu
 *
 *  Created on: Jun 16, 2022
 *      Author: moritz
 */

#include "BoostSolver.h"

using namespace std;

// https://www.boost.org/doc/libs/1_64_0/libs/numeric/odeint/doc/html/boost_numeric_odeint/getting_started/short_example.html

//stateType A = {}
//ptrA = raw_pointer_cast(&A[0])
//for (int i = 0; i < nModes; i++)
//{
//
//}

template<class Tuple>
__host__ __device__
void DerivativeFunctor::operator()(Tuple tuple) // this functor works on tuples of values
{
	// first, unpack the tuple into position values
//	thrust::tuple<T, T, T, T, T, T> q = thrust::get<0>(tuple);
	const T x   = thrust::get<0>(thrust::get<0>(tuple));
	const T y   = thrust::get<1>(thrust::get<0>(tuple));
	const T z   = thrust::get<2>(thrust::get<0>(tuple));
	const T v_x = thrust::get<3>(thrust::get<0>(tuple));
	const T v_y = thrust::get<4>(thrust::get<0>(tuple));
	const T v_z = thrust::get<5>(thrust::get<0>(tuple));
	// Evaluate magnetic field at the position
	T Bx = 0;
	T By = 0;
	T Bz = B0;
	for(int i=0; i<modeCount; i++)
	{
		T kr = ptr_kx[i] * x +  ptr_ky[i] * y + ptr_kz[i] * z; // scalar product of k and pos
		Bx += ptr_Ax[i] * cos(kr + ptr_beta[i]);
		By += ptr_Ay[i] * cos(kr + ptr_beta[i]);
		Bz += ptr_Az[i] * cos(kr + ptr_beta[i]);
	}
	// the differential equation
//	thrust::tuple<T&, T&, T&, T&, T&, T&> dqdt = thrust::get<1>(tuple);
	thrust::get<0>(thrust::get<1>(tuple)) = v_x;							// dx/dt
	thrust::get<1>(thrust::get<1>(tuple)) = v_y;							// dy/dt
	thrust::get<2>(thrust::get<1>(tuple)) = v_z;							// dz/dt
	thrust::get<3>(thrust::get<1>(tuple)) = R_inverse * (v_y*Bz - v_z*By);	// dv_x/dt
	thrust::get<4>(thrust::get<1>(tuple)) = R_inverse * (v_z*Bx - v_x*Bz);	// dv_y/dt
	thrust::get<5>(thrust::get<1>(tuple)) = R_inverse * (v_x*By - v_y*Bx);	// dv_z/dt
};

DerivativeFunctor::DerivativeFunctor(T R_inverse_, T B0_, int modeCount_, const StateType& beta, const StateType& Ax, const StateType& Ay, const StateType& Az, const StateType& kx, const StateType& ky, const StateType& kz)
: R_inverse(R_inverse_), B0(B0_),  modeCount(modeCount_), ptr_beta(thrust::raw_pointer_cast(&beta[0])),
  ptr_Ax(thrust::raw_pointer_cast(&Ax[0])), ptr_Ay(thrust::raw_pointer_cast(&Ay[0])), ptr_Az(thrust::raw_pointer_cast(&Az[0])),
  ptr_kx(thrust::raw_pointer_cast(&kx[0])), ptr_ky(thrust::raw_pointer_cast(&ky[0])), ptr_kz(thrust::raw_pointer_cast(&kz[0]))
{

}

// Derivative Operator //
LorentzForce::LorentzForce(const T R_inverse_, int particleCount_, T B0_, vector<Mode> modes)
: R_inverse(R_inverse_), particleCount(particleCount_), B0(B0_), modeCount(modes.size()), beta(modes.size()), Ax(modes.size()), Ay(modes.size()), Az(modes.size()), kx(modes.size()), ky(modes.size()), kz(modes.size())
{
	// wrap mode parameters to vectors
	Vec betaVec;
	Vec AxVec;
	Vec AyVec;
	Vec AzVec;
	Vec kxVec;
	Vec kyVec;
	Vec kzVec;
	for (int i = 0; i < modes.size(); i++)
	{
		betaVec.push_back(modes[i].beta);
		AxVec.push_back(modes[i].Axi[0]);
		AyVec.push_back(modes[i].Axi[1]);
		AzVec.push_back(modes[i].Axi[2]);
		kxVec.push_back(modes[i].k[0]);
		kyVec.push_back(modes[i].k[1]);
		kzVec.push_back(modes[i].k[2]);
//		cout << "A^2=" <<  to_string(modes[i].Axi[0]*modes[i].Axi[0] + modes[i].Axi[1]*modes[i].Axi[1] + modes[i].Axi[2]*modes[i].Axi[2]) << "k^2=" << to_string(modes[i].k[0]*modes[i].k[0] + modes[i].k[1]*modes[i].k[1] + modes[i].k[2]*modes[i].k[2]) << endl;
	}
	thrust::copy(betaVec.begin(), betaVec.end(), beta.begin());
	thrust::copy(AxVec.begin(), AxVec.end(), Ax.begin());
	thrust::copy(AyVec.begin(), AyVec.end(), Ay.begin());
	thrust::copy(AzVec.begin(), AzVec.end(), Az.begin());
	thrust::copy(kxVec.begin(), kxVec.end(), kx.begin());
	thrust::copy(kyVec.begin(), kyVec.end(), ky.begin());
	thrust::copy(kzVec.begin(), kzVec.end(), kz.begin());
}

//template <class State, class Derivative>
void LorentzForce::operator() (const StateType& q, StateType& dqdt, const T t)
{
	thrust::for_each(
		thrust::make_zip_iterator(thrust::make_tuple(
			// thrust::tuple is limited to a maximum of 10 components.
			// Hence a nested tuple is constructed
			thrust::make_zip_iterator(thrust::make_tuple(
				boost::begin(q) + 0*particleCount, 			// x
				boost::begin(q) + 1*particleCount, 			// y
				boost::begin(q) + 2*particleCount, 			// z
				boost::begin(q) + 3*particleCount, 			// v_x
				boost::begin(q) + 4*particleCount, 			// v_y
				boost::begin(q) + 5*particleCount)), 		// v_z
			thrust::make_zip_iterator(thrust::make_tuple(
				boost::begin(dqdt) + 0*particleCount,		// dx/dt
				boost::begin(dqdt) + 1*particleCount, 		// dy/dt
				boost::begin(dqdt) + 2*particleCount,		// dz/dt
				boost::begin(dqdt) + 3*particleCount,		// dv_x/dt
				boost::begin(dqdt) + 4*particleCount, 		// dv_y/dt
				boost::begin(dqdt) + 5*particleCount)))),	// dv_z/dt
		thrust::make_zip_iterator(thrust::make_tuple(
			thrust::make_zip_iterator(thrust::make_tuple(
				boost::begin(q) + 1*particleCount, 			// x
				boost::begin(q) + 2*particleCount, 			// y
				boost::begin(q) + 3*particleCount, 			// z
				boost::begin(q) + 4*particleCount, 			// v_x
				boost::begin(q) + 5*particleCount, 			// v_y
				boost::begin(q) + 6*particleCount)), 		// v_z
			thrust::make_zip_iterator(thrust::make_tuple(
				boost::begin(dqdt) + 1*particleCount,		// dx/dt
				boost::begin(dqdt) + 2*particleCount, 		// dy/dt
				boost::begin(dqdt) + 3*particleCount,		// dz/dt
				boost::begin(dqdt) + 4*particleCount,		// dv_x/dt
				boost::begin(dqdt) + 5*particleCount, 		// dv_y/dt
				boost::begin(dqdt) + 6*particleCount)))),	// dv_z/dt
		DerivativeFunctor(R_inverse, B0, modeCount, beta, Ax, Ay, Az, kx, ky, kz));
}

// Observer //
Observer::Observer(vector<Vec>& states_, vector<T>& times_)
: states(states_), times(times_)
{ }
void Observer::operator()(const Vec& q, T t)
{
	states.push_back(q);
	times.push_back(t);
}


// Boost Solver //
BoostSolver::BoostSolver(const int particleCount_, const T totalSimulationTime_, const int outputPoints_, const int stepsPerOutput_, const T dt, const T R, const T V_, const FieldGenerator& field, const T omega_, const T Lc_)
: Engine(totalSimulationTime_, outputPoints_, stepsPerOutput_, dt, R, V_, field, omega_), particleCount(particleCount_), omega(omega_), Lc(Lc_)
{
	name = "Runge-Kutta (boost ODEint) Simulation";
}
BoostSolver::BoostSolver(const T totalSimulationTime_, const int outputPoints_, const int stepsPerOutput_, const T dt, const T R, const T V_, const Vec q0_, const FieldGenerator& field, const T omega_)
: Engine(totalSimulationTime_, outputPoints_, stepsPerOutput_, dt, R, V_, q0_, field, omega_), particleCount(1), omega(omega_), Lc(0)
{
	name = "Runge-Kutta (boost ODEint) Simulation";
	cout << endl << "Boost: q0z, R, dt, omega=" << q0_[5] << ", " << R << ", " << dt << ", " << omega_ << endl;
}
void BoostSolver::Randomize_q0()
{
	// Random Engine for initial particle velocity direction
	RandomGen* rg = RandomGen::GetInstance();
	// Randomize starting direction
	T phi = rg->RandomFloat_0_2PI();
	T eta = rg->RandomFloat_m1_1();
	// Generate random direction with v=c
	Vec randomVec(UnitVec(eta, phi));
	Scale(randomVec, V); // Apply speed (scaling of position is irrelevant as position is zero)
	q0 = {0, 0, 0}; // [pc],[pc·Ω] Initial position and velocity of the particle
	AppendVector(q0, randomVec); // Append velocity to position in initial state vector q0
	Scale(q0, V);
}
float BoostSolver::Simulation(vector<Vec>& trajectory_, Vec& time_, int batchNo)
{
	// Initialize simulation
	PrintTime();
	const clock_t beginComputingTime = clock(); // https://stackoverflow.com/questions/728068/how-to-calculate-a-time-difference-in-c
	cout << name << " start" << flush;

	// prepare q0 vector
	Vec q0Vec(6*particleCount);
	//cout << "maxVecSize=" << std::vector<T>::max_size() << endl;
	//cout << "maxdoubleVecSize=" << q0Vec.max_size() << endl;
	//cout << "len(q0Vec)=" << q0Vec.size() << endl;
	for (int n = 0; n < particleCount; n++)
	{
		if (particleCount > 1) // if only one particle is simulated q0 will be set from outside
			Randomize_q0();
		q0Vec[0*particleCount + n] = 0; // x
		q0Vec[1*particleCount + n] = 0; // y
		q0Vec[2*particleCount + n] = 0; // z
		q0Vec[3*particleCount + n] = q0[3]; // v_x
		q0Vec[4*particleCount + n] = q0[4]; // v_y
		q0Vec[5*particleCount + n] = q0[5]; // v_z
	}
	cout << "q=" << q0[0] << ", " << q0[1] << ", " << q0[2] << ", " << q0[3] << ", " << q0[4] << ", " << q0[5] << endl;
	StateType q(6*particleCount);
	//cout << "len(q)=" << q.size() << endl;
	thrust::copy(q0Vec.begin(), q0Vec.end(), q.begin());
	//cout << "len(q)=" << q.size() << endl;
	//cout << "maxStateTypeSize=" << q.max_size() << endl;

	// integrate
	vector<Vec> trajectory;
	Vec time;
	boost::numeric::odeint::runge_kutta4<StateType> stepper;
	LorentzForce lorentzForce(R_inverse, particleCount, field.GetB0(), field.GetModes());
	Observer observer(trajectory, time);

	for (int i = 0; i < outputPoints; i++)
	{
		Vec qVec(6*particleCount);
		//qVec.reserve(6*particleCount);
		thrust::copy(q.begin(), q.end(), qVec.begin());
		observer(qVec, i*dt*stepsPerOutput);
		T time = boost::numeric::odeint::integrate_n_steps(stepper, lorentzForce, q, (T)0., dt, (size_t)stepsPerOutput);//, observer);
		//size_t steps = boost::numeric::odeint::integrate_const(stepper, lorentzForce, q, i*stepsPerOutput*dt, (i+1)*stepsPerOutput*dt, dt);
		//cout << "steps, stepsPerOutput" << steps << ", " << stepsPerOutput << endl;
	}

	// Validate
	assert(trajectory.size() == time.size());
	float timeElapsedInSeconds = float(clock() - beginComputingTime) /  CLOCKS_PER_SEC;
	cout << " finished. Time elapsed: " << timeElapsedInSeconds << " s" << endl;

	// Print
	//vector<Printer*> printers;
	//printers.reserve(particleCount);
	cout << "Writing to files in '" << Printer::GetOutputPath() << "'..." << flush;
	// Instantiate Printers
	for (int particle = 0; particle < particleCount; particle++)
	{
		string filename = "batch" + to_string(batchNo) + "_particle" + to_string(particle) + ".csv";
		string header = "t/(OMEGA^-1); x/Lc; y/Lc; z/Lc; v_x/c, v_y/c, v_z/c";
		T normFac = 1/Lc;
		if (Lc == 0) // This case is for the homogeneous background field, where the correlation length would be infinite
		{
			header = "t/(OMEGA^-1); x/pc; y/pc; z/pc; v_x/c, v_y/c, v_z/c";
			normFac = 1;
		}
		Printer printer(filename, header);
		for (int i = 0; i < outputPoints; i++)
		{
			T t = time[i];
			Vec qVec(trajectory[i]); // Copy construct
			Vec q7Vec;
			q7Vec.push_back(t*omega);
			q7Vec.push_back(qVec[0*particleCount + particle] * normFac);
			q7Vec.push_back(qVec[1*particleCount + particle] * normFac);
			q7Vec.push_back(qVec[2*particleCount + particle] * normFac);
			q7Vec.push_back(qVec[3*particleCount + particle]); // [c]
			q7Vec.push_back(qVec[4*particleCount + particle]); // [c]
			q7Vec.push_back(qVec[5*particleCount + particle]); // [c]
			printer.Write(q7Vec);
		}
	}
	/*for (int particle = 0; particle < particleCount; particle++)
	{
		string filename = "batch" + to_string(batchNo) + "_particle" + to_string(particle) + ".csv";
		string header = "t/(OMEGA^-1); x/Lc; y/Lc; z/Lc; v_x/c, v_y/c, v_z/c";
		printers.push_back(new Printer(filename, header)); // TODO: how to do this without new?
	}
	cout << "len(printers)=" << printers.size() << endl;
	cout << "maxPrinterVecSize=" << printers.max_size() << endl;
	// Print data loop first over points and in the inner loop over particles (and printers) such that all printers Write "in parallel"
	for (int i = 0; i < outputPoints; i++)
	{
		T t = time[i];
		Vec qVec(trajectory[i]); // Copy construct
		for (int particle = 0; particle < particleCount; particle++)
		{
			Vec q7Vec;
			q7Vec.push_back(t*omega);
			q7Vec.push_back(qVec[0*particleCount + particle] / Lc);
			q7Vec.push_back(qVec[1*particleCount + particle] / Lc);
			q7Vec.push_back(qVec[2*particleCount + particle] / Lc);
			q7Vec.push_back(qVec[3*particleCount + particle]); // [c]
			q7Vec.push_back(qVec[4*particleCount + particle]); // [c]
			q7Vec.push_back(qVec[5*particleCount + particle]); // [c]
			printers[particle]->Write(q7Vec);
		}
	}
	// Delete Printers
	for (int particle = 0; particle < particleCount; particle++)
	{
		delete printers[particle];
	}*/
	cout << "finished writing." << endl;
	return timeElapsedInSeconds;
}
void BoostSolver::Step(const T t, Vec& q, Vec& dqdt, Vec& q_out)
{

}

